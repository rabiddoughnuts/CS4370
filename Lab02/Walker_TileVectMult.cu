// Brandon Walker
// CS4370
// Parallel Programming Many-Core GPUs
// Meilin Liu
// 9-Oct-2024
// Tiled Matrix Multiplication

#include <iostream>
#include <hip/hip_runtime.h>
#include <chrono>

using namespace std;

void init_matrix(int *A, int *B, int Width);
void mult_matrix_cpu(int* A, int* B, int* C, int Width);
__global__ void mult_matrix_gpu(int* d_A, int* d_B, int* d_C, int Width);
void compare_matrices(int *cpu_result, int *gpu_result, int Width);
void print_matrix(int *matrix, int Width, const char *name);

int main(){
    int Width, block_size;

    // Get Matrix size from user
    cout << "Enter size of the Width x Width matrix: ";
    cin >> Width;

    // Get bloack size from user
    cout << "Enter the block size for CUDA: ";
    cin >> block_size;

    // Allocate memory for matrices
    int *A = new int[Width * Width];
    int *B = new int[Width * Width];
    int *C_cpu = new int[Width * Width];
    int *C_gpu = new int[Width * Width];

    init_matrix(A, B, Width);

    print_matrix(A, Width, "Matrix A");
    print_matrix(B, Width, "Matrix B");

    auto start_cpu = chrono::high_resolution_clock::now();

    mult_matrix_cpu(A, B, C_cpu, Width);

    auto end_cpu = chrono::high_resolution_clock::now();
    chrono::duration<float, milli> duration_cpu = end_cpu - start_cpu;
    cout << "CPU time: " << duration_cpu.count() << " ms" << endl;

    int *d_A, *d_B, *d_C;
    hipMalloc(&d_A, Width * Width * sizeof(int));
    hipMalloc(&d_B, Width * Width * sizeof(int));
    hipMalloc(&d_C, Width * Width * sizeof(int));

    hipMemcpy(d_A, A, Width * Width * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_B, B, Width * Width * sizeof(int), hipMemcpyHostToDevice);

    dim3 dimBlock(block_size, block_size);
    dim3 dimGrid((Width + block_size - 1) / block_size, (Width + block_size - 1) / block_size);

    hipEvent_t start_gpu, stop_gpu;
    hipEventCreate(&start_gpu);
    hipEventCreate(&stop_gpu);

    hipEventRecord(start_gpu);

    size_t shared_mem_size = 2 * block_size * block_size * sizeof(float);
    mult_matrix_gpu<<<dimGrid, dimBlock, shared_mem_size>>>(d_A, d_B, d_C, Width);

    hipEventRecord(stop_gpu);
    hipEventSynchronize(stop_gpu);

    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start_gpu, stop_gpu);

    cout << "GPU time: " << milliseconds << " ms" << endl;

    hipMemcpy(C_gpu, d_C, Width * Width * sizeof(int), hipMemcpyDeviceToHost);

    print_matrix(C_gpu, Width, "Matrix C (GPU)");

    compare_matrices(C_cpu, C_gpu, Width);

    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);

    delete[] A;
    delete[] B;
    delete[] C_cpu;
    delete[] C_gpu;
    
    return 0;
}

void init_matrix(int *A, int *B, int Width){
    int init=1325;
    for(int row = 0; row < Width; row++){
        for(int col = 0; col < Width; col++){
            init= 3125 * init % 6553;
            A[row * Width + col] = (init - 1000) / 6553;
            B[row * Width + col] = init % 251;
        }
    }
}

/**
void init_matrix(int *A, int *B, int Width){
    int init=1325;
    for(int row = 0; row < Width; row++){
        for(int col = 0; col < Width; col++){
            init= (3125 * init) % 65536;
            A[row * Width + col] = (init - 32768) / 6553;
            B[row * Width + col] = init % 1000;
        }
    }
}
**/

void mult_matrix_cpu(int *A, int *B, int *C, int Width){
    for(int row = 0; row < Width; row++){
        for(int col = 0; col < Width; col++){
            int sum = 0;
            for(int k = 0; k < Width; k++){
                int m = A[row * Width + k];
                int n = B[k * Width + col];
                sum += m * n;
            }
            C[row * Width + col] = sum;
        }
    }
}

__global__ void mult_matrix_gpu(int* d_A, int* d_B, int* d_C, int Width){
    extern __shared__ int shared_mem[];
    int* ds_A = shared_mem;
    int* ds_B = shared_mem + blockDim.x * blockDim.y;

    int bx = blockIdx.x;
    int by = blockIdx.y;
    int tx = threadIdx.x;
    int ty = threadIdx.y;

    int row = by * blockDim.y + ty;
    int col = bx * blockDim.x + tx;

    int P = 0;

    for (int k = 0; k < (Width + blockDim.x - 1) / blockDim.x; ++k) {
        if (row < Width && k * blockDim.x + tx < Width) {
            ds_A[ty * blockDim.x + tx] = d_A[row * Width + k * blockDim.x + tx];
        } else {
            ds_A[ty * blockDim.x + tx] = 0;
        }
        if (col < Width && k * blockDim.x + ty < Width) {
            ds_B[ty * blockDim.x + tx] = d_B[(k * blockDim.x + ty) * Width + col];
        } else {
            ds_B[ty * blockDim.x + tx] = 0;
        }

        __syncthreads();
        
        for (int n = 0; n < blockDim.x; ++n) {
            P += ds_A[ty * blockDim.x + n] * ds_B[n * blockDim.x + tx];
        }

        __syncthreads();
    }

    if (row < Width && col < Width) {
        d_C[row * Width + col] = P;
    }
}

void compare_matrices(int *cpu_result, int *gpu_result, int Width){
    for(int row = 0; row < Width; row++){
        for(int col = 0; col < Width; col++){
            int idx = row * Width + col;
            if(cpu_result[idx] != gpu_result[idx]){
                cout << "Mismatch at index (" << row << ", " << col << ")! CPU: " << cpu_result[idx] << ", GPU: " << gpu_result[idx] << endl;
                return;
            }
        }
        
    }
    cout << "CPU and GPU results match!" << endl;
}

void print_matrix(int *matrix, int Width, const char *name){
    cout << name << ":" << endl;
    if (Width <= 20){
        for(int row = 0; row < Width; row++){
            for(int col = 0; col < Width; col++){
                cout << matrix[row * Width + col] << " ";
            }
            cout << endl;
        }
    } else {
        cout << "Matrix too large, printing only the first row:" << endl;
        for(int col = 0; col < Width; col++){
            cout << matrix[col] << " ";
        }
        cout << endl;
    }
    cout << endl;
}
