#include "hip/hip_runtime.h"
// Brandon Walker
// CS4370
// Parallel Programming Many-Core GPUs
// Meilin Liu
// 23-Oct-2024
// Tiled Matrix Multiplication

#include <iostream>
#include <hip/hip_runtime.h>
#include <chrono>

using namespace std;

void init_matrix(int *A, int *B, int N);
void SumReduction(int* x, int N);
__global__ void SumReductionKernel(int* x, int N);
void compare_matrices(int *cpu_result, int *gpu_result, int N);
void print_matrix(int *matrix, int N, const char *name);

int main(){
    int Width, block_size;

    // Get Matrix size from user
    cout << "Enter size of the Width x Width matrix: ";
    cin >> Width;

    // Get bloack size from user
    cout << "Enter the block size for CUDA: ";
    cin >> block_size;

    // Allocate memory for matrices
    int *A = new int[Width * Width];
    int *B = new int[Width * Width];

    init_matrix(A, B, Width);

    print_matrix(A, Width, "Matrix A");
    print_matrix(B, Width, "Matrix B");

    auto start_cpu = chrono::high_resolution_clock::now();

    mult_matrix_cpu(A, B, C_cpu, Width);

    auto end_cpu = chrono::high_resolution_clock::now();
    chrono::duration<float, milli> duration_cpu = end_cpu - start_cpu;
    cout << "CPU time: " << duration_cpu.count() << " ms" << endl;

    int *d_B;
    hipMalloc(&d_B, Width * Width * sizeof(int));

    hipMemcpy(d_B, B, Width * Width * sizeof(int), hipMemcpyHostToDevice);

    dim3 dimBlock(block_size, block_size);
    dim3 dimGrid((Width + block_size - 1) / block_size, (Width + block_size - 1) / block_size);

    hipEvent_t start_gpu, stop_gpu;
    hipEventCreate(&start_gpu);
    hipEventCreate(&stop_gpu);

    hipEventRecord(start_gpu);

    size_t shared_mem_size = 2 * block_size * block_size * sizeof(float);
    SumReductionKernel<<<dimGrid, dimBlock, shared_mem_size>>>(d_B, Width);

    hipEventRecord(stop_gpu);
    hipEventSynchronize(stop_gpu);

    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start_gpu, stop_gpu);

    cout << "GPU time: " << milliseconds << " ms" << endl;

    hipMemcpy(B, d_B, Width * Width * sizeof(int), hipMemcpyDeviceToHost);

    print_matrix(A, Width, "Matrix A (CPU)");
    print_matrix(B, Width, "Matrix B (GPU)");

    compare_matrices(A, B, Width);

    hipFree(d_B);

    delete[] A;
    delete[] B;
    
    return 0;
}

void init_matrix(int *A, int *B, int N){
    int init = 1325;
    for(int i = 0; i < N; i++){
        init = 3125 * init % 6553;
        A[i] = (init - 1000) % 97;
        B[i] = (init - 1000) % 97;
    }
}

void SumReduction(int* x, int N){
    for(int i = 1; i < N; i++){
        x[0] += x[i];
    }
    int overallSum = x[0];
    return overallSum;
}

__global__ void SumReductionKernel(int* x, int N){
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int half = N / 2;
    while(half > 0){
        if(i < half){
            x[i] += x[i + half];
        }
        __syncthreads();
        half /= 2;
    }
}

void compare_matrices(int *cpu_result, int *gpu_result, int N){
    for(int i = 0; i < N; i++){
        if(cpu_result[i] != gpu_result[i]){
            cout << "Matrices are not equal" << endl;
            return;
        }
    }
    cout << "Matrices are equal" << endl;
}

void print_matrix(int *matrix, int N, const char *name){
    cout << name << ":" << endl;
    for(int i = 0; i < N; i++){
        cout << matrix[i] << " ";
        if((i + 1) % 10 == 0){
            cout << endl;
        }
    }
    cout << endl;
}